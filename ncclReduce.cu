#include "hip/hip_runtime.h"
/*%****************************************************************************80
%  Code: 
%   ncclReduce.cu
%
%  Purpose:
%   Implements sample reduce code using the package NCCL(ncclReduce).
%   Using 'Multiples Devices per Thread'.
%   Implements dot product(scalar product).
%   x = (xo, x1, x2, ..., xn)
%   y = (yo, y1, y2, ..., yn)
%   c = (xo . yo + x1 . y1 + ..., xn . yn)
%
%  Modified:
%   Aug 18 2020 10:57 
%
%  Author:
%   Murilo Boratto <murilo.boratto 'at' fieb.org.br>
%
%  How to Compile:
%   nvcc ncclReduce.cu -o ncclReduce -lnccl  
%
%  Execute: 
%   ./ncclReduce                           
%   
%****************************************************************************80*/
#include <unistd.h>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>
#include "/home/rogerlee/code/nccl/build/include/nccl.h"
#include <math.h>
const int ngpu = 8;


void NCCLInit(ncclComm_t* &comm, hipStream_t* &s, int* &device){
  comm = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * ngpu);  
  s    = (hipStream_t*)malloc(sizeof(hipStream_t)* ngpu);
  device = (int *)malloc(ngpu*sizeof(int));

  for(int g=0;g<ngpu;g++){
    device[g] = g;
    hipSetDevice(device[g]); 
    hipStreamCreate(&s[g]);
    hipDeviceSynchronize();  
    hipStreamSynchronize(s[g]);
  }
  ncclCommInitAll(comm, ngpu, device);

}

template<class T>
void CudaMemInit(T** &x, const long long data_size, int* device){
  x  = (T**)malloc(ngpu*sizeof(T*));
  for(int g=0;g<ngpu;g++){
    hipSetDevice(device[g]);
    hipMalloc(&x[g], data_size * sizeof(T));
  }
}

void ncclfinish(ncclComm_t* &comm, hipStream_t* &s, int* device){
    for(int g = 0; g < ngpu; g++) { /*Destroy CUDA Streams*/
        hipSetDevice(device[g]);
        hipStreamDestroy(s[g]);
    }

    for(int g = 0; g < ngpu; g++) /*Finalizing NCCL*/
      ncclCommDestroy(comm[g]);
}

template<class T>
float allreduceTest(long long data_size, int* device, int loop, ncclComm_t* comm, hipStream_t* s, T** &src, T** &dst, size_t mode){
  int cnt = 0;
  float elapsedTime = 0.0, totaltime = 0.0;
  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);

  // hipEvent_t e_start[ngpu];
  // hipEvent_t e_stop[ngpu];
  // for(int g=0;g<ngpu;g++){
  //   if(hipEventCreate(&e_start[g])!=hipSuccess){
  //     printf("Create %d start event ERROR\n",g);
  //     return;
  //   }
  //   if(hipEventCreate(&e_stop[g])!=hipSuccess){
  //     printf("Create %d stop event ERROR\n",g);
  //     return;
  //   }      
  // }
    
  
  while(cnt<loop){
    ncclGroupStart(); 
    for(int g = 0; g < ngpu; g++) {
      hipSetDevice(device[g]);
      // ncclAllReduce(src[g], dst[g], data_size, ncclDouble, ncclSum, comm[g], s[g], mode); 
      // if(split == 0){
      //   ncclAllReduce(src[g], dst[g], data_size, ncclDouble, ncclSum, comm[g], s[g], mode); 
      // }else{
        int offset = data_size/2;
        ncclAllReduce(src[g], dst[g], offset, ncclDouble, ncclSum, comm[g], 0, 3); 
        ncclAllReduce(src[g]+offset, dst[g]+offset, data_size - offset, ncclDouble, ncclSum, comm[g], 0, 2); 
      // }
    }
    ncclGroupEnd();
    ncclGroupStart(); 
    for(int g = 0; g < ngpu; g++) {
      hipSetDevice(device[g]);
      // ncclAllReduce(src[g], dst[g], data_size, ncclDouble, ncclSum, comm[g], s[g], mode); 
      // if(split == 0){
      //   ncclAllReduce(src[g], dst[g], data_size, ncclDouble, ncclSum, comm[g], s[g], mode); 
      // }else{
        int offset = data_size/2;
        ncclAllReduce(src[g], dst[g], offset, ncclDouble, ncclSum, comm[g], 0, 2); 
        ncclAllReduce(src[g]+offset, dst[g]+offset, data_size - offset, ncclDouble, ncclSum, comm[g], 0, 3); 
      // }
    }
    // for(int g=0;g<ngpu;g++){
    //   hipEventRecord(e_start[g], s[g]);  
    // }
    hipEventRecord(e_start, 0);
    ncclGroupEnd();

    for(int g = 0; g < ngpu; g++) {
        hipSetDevice(device[g]);    
        hipDeviceSynchronize();  
        hipStreamSynchronize(s[g]);     
        // hipEventRecord(e_stop[g], s[g]);   
        // hipEventSynchronize(e_stop[g]);
        // hipEventElapsedTime(&totaltime, e_start[g], e_stop[g]);
        // elapsedTime = elapsedTime+totaltime;
    }
    hipEventRecord(e_stop, 0);
    hipEventSynchronize(e_stop);
    hipEventElapsedTime(&totaltime, e_start, e_stop);
    elapsedTime = elapsedTime+totaltime;
    cnt++;
  }
  // for(int g=0;g<ngpu;g++){
  //   hipEventDestroy(e_start[g]);
  //   hipEventDestroy(e_stop[g]); 
  // }
  hipEventDestroy(e_start);
  hipEventDestroy(e_stop);
  return elapsedTime/8;
  
}

template<class T>
long long CheckError(T* x, long long data_size){
  long long cnt = 0;
  // int a;
  for(long long i=0;i<data_size;i++){
    if(x[i]!=(T)(i*ngpu)){
      cnt++;
      // printf("%lld %lf\n", i, x[i]);
      // scanf("%d", &a);
    }
  }
  return cnt;
}

template<class T>
void GenData(T* &x, long long data_size){
  x   = (T*)malloc(sizeof(T)*data_size);
  for(long long i = 0; i < data_size; i++){ 
    x[i] = (T)(i);
  }
}

int main(int argc, char* argv[]) {

  int loop = -1, split =-1;
  if(argc == 1){
    printf("Argument Error\n");
    return 1;
  }
  if(argc > 1)loop = atoi(argv[1]);
  if(argc > 2)split = 1;
  double *x;
  // 4,999,900,001 2,147,483,648
  long long data_size = 2147483648;//16GB
  GenData(x,data_size);
  ncclComm_t* comm;
  hipStream_t* s;
  int* device;
  NCCLInit(comm, s, device);
  printf("ncclInit\n");
  double**device_src = nullptr, **device_dst = nullptr;
  CudaMemInit(device_src, data_size, device);
  CudaMemInit(device_dst, data_size, device);
  printf("CudaMemInit\n");
  for(int g=0;g<ngpu;g++){
    hipSetDevice(device[g]);
    hipMemcpy(device_src[g],  x, data_size * sizeof(double), hipMemcpyHostToDevice); /*Copy from Host to Devices*/
  }

  float time;
  size_t mode = 0;
  clock_t begin, end;

  // printf("******   warm up mode = %d ******\n", mode);
  // allreduceTest(data_size, device, 5, comm, s, device_src, device_dst, mode);
  // printf("******************************\n");

  // begin = clock();
  // time = allreduceTest(data_size, device, loop, comm, s, device_src, device_dst, mode);
  // end = clock();
  // hipSetDevice(device[0]);
  // hipMemcpy(x,  device_dst[0], data_size * sizeof(double), hipMemcpyDeviceToHost);
  // printf("Total ERROR %lld\n", CheckError(x, data_size));
  // printf("kernel cost %3.1f ms\n", time);
  // printf("Total time %lf s\n", (double)(end - begin) / CLOCKS_PER_SEC);
  // printf("******************************\n\n");


  // mode = 1;
  // printf("******   warm up mode = %d ******\n", mode);
  // allreduceTest(data_size, device, 5, comm, s, device_src, device_dst, mode);
  // printf("******************************\n");

  // begin = clock();
  // time = allreduceTest(data_size, device, loop, comm, s, device_src, device_dst, mode);
  // end = clock();
  // hipSetDevice(device[0]);
  // hipMemcpy(x,  device_dst[0], data_size * sizeof(double), hipMemcpyDeviceToHost);
  // printf("Total ERROR %lld\n", CheckError(x, data_size));
  // printf("kernel cost %3.1f ms\n", time);
  // printf("Total time %lf s\n", (double)(end - begin) / CLOCKS_PER_SEC);
  // printf("******************************\n\n");

  mode = 7;
  printf("******   warm up mode = %zu ******\n", mode);
  allreduceTest(data_size, device, 5, comm, s, device_src, device_dst, mode);
  printf("******************************\n");

  begin = clock();
  time = allreduceTest(data_size, device, loop, comm, s, device_src, device_dst, mode);
  end = clock();
  hipSetDevice(device[0]);
  hipMemcpy(x,  device_dst[0], data_size * sizeof(double), hipMemcpyDeviceToHost);
  printf("Total ERROR %lld\n", CheckError(x, data_size));
  printf("kernel cost %3.1f ms\n", time);
  printf("Total time %lf s\n", (double)(end - begin) / CLOCKS_PER_SEC);
  printf("******************************\n\n");



  ncclfinish(comm, s, device);
  hipFree(device_src);
  hipFree(device_dst);
  return 0;

}/*main*/




